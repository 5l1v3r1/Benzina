#include "hip/hip_runtime.h"
/* Includes */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "benzina/benzina.h"
#include "kernels.h"


/* Defines */



/* CUDA kernels */

/**
 * @brief CUDA post-processing kernel
 * @param x
 */

BENZINA_PLUGIN_HIDDEN __global__ void nvdecodePostprocKernel(float* x){
	
}

extern "C" BENZINA_PLUGIN_HIDDEN void nvdecodePostprocKernelInvoker(void){
	dim3 Dg = {1,1,1}, Db = {1,1,1};
	hipStream_t stream;
	hipStreamCreate(&stream);
	nvdecodePostprocKernel<<<Dg, Db, 0, stream>>>(NULL);
}
