#include "hip/hip_runtime.h"
/* Includes */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "benzina/benzina.h"
#include "kernels.h"


/* Defines */



/* CUDA kernels */

/**
 * @brief CUDA post-processing kernel
 * @param x
 */

BENZINA_PLUGIN_HIDDEN __global__ void nvdecodePostprocKernel(float* x){
	
}

BENZINA_PLUGIN_HIDDEN int   nvdecodePostprocKernelInvoker(hipStream_t cudaStream,
                                                          void*        dstPtr,
                                                          unsigned     dstH,
                                                          unsigned     dstW,
                                                          float        OOB0,
                                                          float        OOB1,
                                                          float        OOB2,
                                                          float        B0,
                                                          float        B1,
                                                          float        B2,
                                                          float        S0,
                                                          float        S1,
                                                          float        S2,
                                                          float        H00,
                                                          float        H01,
                                                          float        H02,
                                                          float        H10,
                                                          float        H11,
                                                          float        H12,
                                                          float        H20,
                                                          float        H21,
                                                          float        H22,
                                                          unsigned     colorMatrix,
                                                          void*        srcPtr,
                                                          unsigned     srcPitch,
                                                          unsigned     srcH,
                                                          unsigned     srcW){
	dim3 Dg = {1,1,1}, Db = {1,1,1};
	nvdecodePostprocKernel<<<Dg, Db, 0, cudaStream>>>((float*)dstPtr);
	return 0;
}
